#include "hip/hip_runtime.h"
%% writefile hello.cu

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void findShortestSides(int* d_A, int* d_B, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {

        int a = d_A[3 * idx];
        int b = d_A[3 * idx + 1];
        int c = d_A[3 * idx + 2];


        d_B[idx] = min(a, min(b, c));
    }
}


__global__ void findMaxSide(int* d_B, int* d_maxSide, int n) {
    __shared__ int shared_max[256];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadIdx.x;


    shared_max[threadId] = (idx < n) ? d_B[idx] : INT_MIN;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadId < stride && idx + stride < n) {
            shared_max[threadId] = max(shared_max[threadId], shared_max[threadId + stride]);
        }
        __syncthreads();
    }

    if (threadId == 0) {
        atomicMax(d_maxSide, shared_max[0]);
    }
}

int main() {
    int n;
    cout << "Unesi broj trouglova: ";
    cin >> n;

    int* A = new int[3 * n]; 
    int* B = new int[n];     


    cout << "Unesi stranice trouglova: ";
    for (int i = 0; i < 3 * n; i++) {
        cin >> A[i];
    }


    int* d_A, * d_B, * d_maxSide;
    hipMalloc(&d_A, 3 * n * sizeof(int));
    hipMalloc(&d_B, n * sizeof(int));
    hipMalloc(&d_maxSide, sizeof(int));

    hipMemcpy(d_A, A, 3 * n * sizeof(int), hipMemcpyHostToDevice);


    int maxSide = 0;
    hipMemcpy(d_maxSide, &maxSide, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

  
    findShortestSides << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, n);

    findMaxSide << <blocksPerGrid, threadsPerBlock >> > (d_B, d_maxSide, n);

    hipMemcpy(&maxSide, d_maxSide, sizeof(int), hipMemcpyDeviceToHost);


    cout << "Najveca od najkracih stranica je: " << maxSide << endl;


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_maxSide);
    delete[] A;
    delete[] B;

    return 0;
}

