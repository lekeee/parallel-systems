#include "hip/hip_runtime.h"
%%writefile hello.cu

#include <iostream>
#include <hip/hip_runtime.h>

#define n 3
#define PI 3.14159

struct Kruznica {
    int xi;
    int yi;
    int r;
};

__global__ void findMax(Kruznica *d_array, int *d_maxIdx) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float sharedAreas[128]; 
    __shared__ int sharedIdx[128];    

    if (tid < n) {
        sharedAreas[threadIdx.x] = PI * d_array[tid].r * d_array[tid].r;
        sharedIdx[threadIdx.x] = tid;
    } else {
        sharedAreas[threadIdx.x] = 0;
        sharedIdx[threadIdx.x] = -1;
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride && sharedAreas[threadIdx.x] < sharedAreas[threadIdx.x + stride]) {
            sharedAreas[threadIdx.x] = sharedAreas[threadIdx.x + stride];
            sharedIdx[threadIdx.x] = sharedIdx[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicMax(d_maxIdx, sharedIdx[0]);
    }
}

int main() {

    Kruznica array[n] = {
        {1, 2, 3},
        {4, 5, 4},
        {10, 1, 1}
    };

    Kruznica *d_array;
    int *d_maxIdx;
    int maxIdx = 0;

    size_t arrayBytes = n * sizeof(Kruznica);

    hipMalloc(&d_array, arrayBytes);
    hipMalloc(&d_maxIdx, sizeof(int));


    hipMemcpy(d_array, array, arrayBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_maxIdx, &maxIdx, sizeof(int), hipMemcpyHostToDevice);

    int threadNum = 128;
    int blockNum = (n + threadNum - 1) / threadNum;

    findMax<<<blockNum, threadNum>>>(d_array, d_maxIdx);

    hipMemcpy(&maxIdx, d_maxIdx, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Kruznica sa najvecom povrsinom: \n";
    std::cout << "Centar: (" << array[maxIdx].xi << ", " << array[maxIdx].yi << ")\n";
    std::cout << "Poluprecnik: " << array[maxIdx].r << "\n";
    std::cout << "Povrsina: " << PI * array[maxIdx].r * array[maxIdx].r << "\n";

    hipFree(d_array);
    hipFree(d_maxIdx);

    return 0;
}
